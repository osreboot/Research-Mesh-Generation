#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "math.cuh"

__global__ void circlesNTensorLightweight(bool *out, const float* __restrict__ d_D, const double* __restrict__ dpxy2,
                                    const double comp, const int pointsSize, const int step){
    unsigned int i = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y) * blockDim.x + threadIdx.x;
    for(; i < pointsSize; i += step){
        out[i] = static_cast<double>(d_D[i]) + dpxy2[i] <= comp;
    }
}

// Simplified distance single-GEMM Tensor Core algorithm (WITHOUT Tensor Core invocation for metrics gathering purposes)
class CirclesNTensorLightweight : public Circles{

private:
    int n = 0, m = 0, k = 0;

    float *px = nullptr, *py = nullptr, *ident = nullptr;
    int pointsSize = 0;
    double *pxy2 = nullptr, *dpxy2 = nullptr;
    bool *doutput = nullptr;

    float *d_C = nullptr;

public:
    __host__ void load(const double *pxArg, const double *pyArg, int pointsSizeArg) override {
        pointsSize = pointsSizeArg;

        m = TENSOR_GET_M;
        n = TENSOR_GET_N;
        k = TENSOR_GET_K;

        px = new float[m * k];
        py = new float[m * n];
        pxy2 = new double[pointsSize];

        for(int i = 0; i < m * k; i++){
            if(i < pointsSize){
                px[i] = (float)pxArg[i];
                py[i] = (float)pyArg[i];
                pxy2[i] = pxArg[i] * pxArg[i] + pyArg[i] * pyArg[i];
            }else{
                px[i] = 0.0f;
                py[i] = 0.0f;
            }
        }

        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C), m * n * sizeof(d_C[0])));
        CUBLAS_CHECK(hipblasSetVector(n * m, sizeof(py[0]), py, 1, d_C, 1));

        CUDA_CHECK(hipMalloc((void**)&dpxy2, sizeof(double) * pointsSize));
        CUDA_CHECK(hipMemcpy(dpxy2, pxy2, sizeof(double) * pointsSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc((void**)&doutput, sizeof(bool) * pointsSize));
    }

    __host__ void run(bool *output, const Point& p1, const Point& p2, const Point& p3) override {
        const Circumcircle circle(p1, p2, p3);
        const float coefPx = -2.0f * (float)circle.x;
        const float coefPy = -2.0f * (float)circle.y;
        const double comp = circle.r2 - circle.x * circle.x - circle.y * circle.y + RADIUS_ERROR;

        // Tensor core instructions would be invoked here, to theoretically save compute time for the kernel

        dim3 dimGrid = KERNEL_DIM_GRID;
        dim3 dimBlock = KERNEL_DIM_BLOCK;
        const int step = dimGrid.x * dimGrid.y * dimGrid.z * dimBlock.x;

        circlesNTensorLightweight<<<dimGrid, dimBlock>>>(doutput, d_C, dpxy2, comp, pointsSize, step);
        CUDA_CHECK_LAST_ERROR();
    }

    __host__ void save(bool *output) override {
        CUDA_CHECK(hipMemcpy(output, doutput, sizeof(bool) * pointsSize, hipMemcpyDeviceToHost));
    }

    __host__ void cleanup() override {
        hipFree(d_C);

        hipFree(dpxy2);
        hipFree(doutput);

        delete[] px;
        delete[] py;
        delete[] pxy2;
        delete[] ident;
    }

    string getFileName() const override {
        return "ntensor_lightweight";
    }

};
