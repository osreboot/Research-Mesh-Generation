#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "math.cuh"

__global__ void circlesKernelLightweight(bool *out, const double* __restrict__ dpx, const double* __restrict__ dpy, const double* __restrict__ dpxy2,
                               const double coefPx, const double coefPy, const double comp, const int pointsSize, const int step){
    unsigned int i = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y) * blockDim.x + threadIdx.x;
    for(; i < pointsSize; i += step){
        out[i] = coefPx * dpx[i] + coefPy * dpy[i] + dpxy2[i] <= comp;
    }
}

// Simplified distance-based parallel algorithm
class CirclesKernelLightweight : public Circles{

private:
    const double *px = nullptr, *py = nullptr;
    int pointsSize = 0;
    double *dpx = nullptr, *dpy = nullptr, *pxy2 = nullptr, *dpxy2 = nullptr;
    bool *doutput = nullptr;

public:
    __host__ void load(const double *pxArg, const double *pyArg, int pointsSizeArg) override {
        px = pxArg;
        py = pyArg;
        pointsSize = pointsSizeArg;

        pxy2 = new double[pointsSize];

        for(int i = 0; i < pointsSize; i++){
            pxy2[i] = px[i] * px[i] + py[i] * py[i];
        }

        CUDA_CHECK(hipMalloc((void**)&dpx, sizeof(double) * pointsSize));
        CUDA_CHECK(hipMalloc((void**)&dpy, sizeof(double) * pointsSize));
        CUDA_CHECK(hipMalloc((void**)&dpxy2, sizeof(double) * pointsSize));
        CUDA_CHECK(hipMemcpy(dpx, px, sizeof(double) * pointsSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dpy, py, sizeof(double) * pointsSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dpxy2, pxy2, sizeof(double) * pointsSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc((void**)&doutput, sizeof(bool) * pointsSize));
    }

    __host__ void run(bool *output, const Point& p1, const Point& p2, const Point& p3) override {
        const Circumcircle circle(p1, p2, p3);
        const double coefPx = -2.0 * circle.x;
        const double coefPy = -2.0 * circle.y;
        const double comp = circle.r2 - circle.x * circle.x - circle.y * circle.y + RADIUS_ERROR;

        dim3 dimGrid = KERNEL_DIM_GRID;
        dim3 dimBlock = KERNEL_DIM_BLOCK;
        const int step = dimGrid.x * dimGrid.y * dimGrid.z * dimBlock.x;

        circlesKernelLightweight<<<dimGrid, dimBlock>>>(doutput, dpx, dpy, dpxy2, coefPx, coefPy, comp, pointsSize, step);
        CUDA_CHECK_LAST_ERROR();
    }

    __host__ void save(bool *output) override {
        CUDA_CHECK(hipMemcpy(output, doutput, sizeof(bool) * pointsSize, hipMemcpyDeviceToHost));
    }

    __host__ void cleanup() override {
        hipFree(dpx);
        hipFree(dpy);
        hipFree(dpxy2);
        hipFree(doutput);

        delete[] pxy2;
    }

    string getFileName() const override {
        return "kernel_lightweight";
    }

};
