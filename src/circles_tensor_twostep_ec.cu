#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "math.cuh"

__global__ void circlesTensorTwostepEC(bool *out, const double* __restrict__ dpx, const double* __restrict__ dpy, const double* __restrict__ dpxy2,
                                       const double doubleCoefPx, const double doubleCoefPy,
                                       const float* __restrict__ d_D, const double comp, const int pointsSize, const int step){
    unsigned int i = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y) * blockDim.x + threadIdx.x;
    for(; i < pointsSize; i += step){
        const double diff = comp - (double)d_D[i];
        if(fabs(diff) < 0.000001){
            out[i] = doubleCoefPx * dpx[i] + doubleCoefPy * dpy[i] + dpxy2[i] <= comp;
        }else out[i] = diff >= 0.0;
    }
}

// Simplified distance single-GEMM Tensor Core algorithm with precision error correction
class CirclesTensorTwostepEC : public Circles{

private:
    int n = 0, m = 0, k = 0;

    float *px = nullptr, *py = nullptr, *ident = nullptr;
    int pointsSize = 0;
    float *pxy2 = nullptr;
    const double *doubleDPx = nullptr, *doubleDPy = nullptr;
    double *dpx, *dpy, *doublePxy2 = nullptr, *dpxy2 = nullptr;
    bool *doutput = nullptr;

    hipblasHandle_t handle;
    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr, *d_D = nullptr;

public:
    __host__ void load(const double *pxArg, const double *pyArg, int pointsSizeArg) override {
        pointsSize = pointsSizeArg;

        m = TENSOR_GET_M;
        n = TENSOR_GET_N;
        k = TENSOR_GET_K;

        px = new float[m * k];
        py = new float[m * n];
        pxy2 = new float[m * k];

        doubleDPx = pxArg;
        doubleDPy = pyArg;
        doublePxy2 = new double[pointsSize];

        for(int i = 0; i < m * k; i++){
            if(i < pointsSize){
                px[i] = (float)pxArg[i];
                py[i] = (float)pyArg[i];
                pxy2[i] = (float)pxArg[i] * (float)pxArg[i] + (float)pyArg[i] * (float)pyArg[i];
                doublePxy2[i] = pxArg[i] * pxArg[i] + pyArg[i] * pyArg[i];
            }else{
                px[i] = 0.0f;
                py[i] = 0.0f;
                pxy2[i] = 0.0f;
            }
        }

        ident = new float[k * n];
        for(int x = 0; x < n; x++){
            for(int y = 0; y < k; y++){
                ident[y * n + x] = (x == y) ? 1.0f : 0.0f;
            }
        }

        CUBLAS_CHECK(hipblasCreate(&handle));
        CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), m * k * sizeof(d_A[0])));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B), k * n * sizeof(d_B[0])));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C), m * n * sizeof(d_C[0])));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_D), m * n * sizeof(d_D[0])));

        CUBLAS_CHECK(hipblasSetVector(m * k, sizeof(px[0]), px, 1, d_A, 1));
        CUBLAS_CHECK(hipblasSetVector(k * n, sizeof(ident[0]), ident, 1, d_B, 1));
        CUBLAS_CHECK(hipblasSetVector(n * m, sizeof(py[0]), py, 1, d_C, 1));
        CUBLAS_CHECK(hipblasSetVector(n * m, sizeof(pxy2[0]), pxy2, 1, d_D, 1));

        CUDA_CHECK(hipMalloc((void**)&dpx, sizeof(double) * pointsSize));
        CUDA_CHECK(hipMalloc((void**)&dpy, sizeof(double) * pointsSize));
        CUDA_CHECK(hipMemcpy(dpx, doubleDPx, sizeof(double) * pointsSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dpy, doubleDPy, sizeof(double) * pointsSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc((void**)&dpxy2, sizeof(double) * pointsSize));
        CUDA_CHECK(hipMemcpy(dpxy2, doublePxy2, sizeof(double) * pointsSize, hipMemcpyHostToDevice));

        CUDA_CHECK(hipMalloc((void**)&doutput, sizeof(bool) * pointsSize));
    }

    __host__ void run(bool *output, const Point& p1, const Point& p2, const Point& p3) override {
        const Circumcircle circle(p1, p2, p3);
        const float coefPx = -2.0f * (float)circle.x;
        const float coefPy = -2.0f * (float)circle.y;
        const double doubleCoefPx = -2.0 * circle.x;
        const double doubleCoefPy = -2.0 * circle.y;
        const double comp = circle.r2 - circle.x * circle.x - circle.y * circle.y + RADIUS_ERROR;

        CUBLAS_CHECK(hipblasSetVector(n * m, sizeof(py[0]), py, 1, d_C, 1));
        CUBLAS_CHECK(hipblasSetVector(n * m, sizeof(pxy2[0]), pxy2, 1, d_D, 1));

        hipDeviceSynchronize();

        CUBLAS_CHECK(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &coefPx,
                                  d_A, HIP_R_32F, m,
                                  d_B, HIP_R_32F, k, &coefPy,
                                  d_C, HIP_R_32F, m,
                                  HIP_R_32F,
                                  CUBLAS_GEMM_DEFAULT_TENSOR_OP));

        const float alpha = 1.0f;
        const float beta = 1.0f;

        hipDeviceSynchronize();

        CUBLAS_CHECK(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
                                  d_C, HIP_R_32F, m,
                                  d_B, HIP_R_32F, k, &beta,
                                  d_D, HIP_R_32F, m,
                                  HIP_R_32F,
                                  CUBLAS_GEMM_DEFAULT_TENSOR_OP));

        dim3 dimGrid = KERNEL_DIM_GRID;
        dim3 dimBlock = KERNEL_DIM_BLOCK;
        const int step = dimGrid.x * dimGrid.y * dimGrid.z * dimBlock.x;

        hipDeviceSynchronize();

        circlesTensorTwostepEC<<<dimGrid, dimBlock>>>(doutput, dpx, dpy, dpxy2, doubleCoefPx, doubleCoefPy, d_D, comp, pointsSize, step);
        CUDA_CHECK_LAST_ERROR();
    }

    __host__ void save(bool *output) override {
        CUDA_CHECK(hipMemcpy(output, doutput, sizeof(bool) * pointsSize, hipMemcpyDeviceToHost));
    }

    __host__ void cleanup() override {
        CUBLAS_CHECK(hipblasDestroy(handle));

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        hipFree(d_D);

        hipFree(dpx);
        hipFree(dpy);
        hipFree(dpxy2);

        hipFree(doutput);

        delete[] px;
        delete[] py;
        delete[] pxy2;
        delete[] doublePxy2;
        delete[] ident;
    }

    string getFileName() const override {
        return "tensor_twostep";
    }

};
