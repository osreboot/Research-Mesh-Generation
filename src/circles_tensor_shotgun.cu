#include "hip/hip_runtime.h"
#pragma once

#include <mma.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "math.cuh"

#define CUDA_CHECK(err)                                                                            \
    do {                                                                                           \
        hipError_t err_ = (err);                                                                  \
        if (err_ != hipSuccess) {                                                                 \
            std::printf("CUDA error %d at %s:%d\n", err_, __FILE__, __LINE__);                     \
            throw std::runtime_error("CUDA error");                                                \
        }                                                                                          \
    } while (0)

#define CUBLAS_CHECK(err)                                                                          \
    do {                                                                                           \
        hipblasStatus_t err_ = (err);                                                               \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                                       \
            std::printf("cublas error %d at %s:%d\n", err_, __FILE__, __LINE__);                   \
            throw std::runtime_error("cublas error");                                              \
        }                                                                                          \
    } while (0)

__global__ void circlesAssignOutput(bool *out, const float* d_D, const double* __restrict__ dpxy2, const double comp, int pointsSize, int threads){
    for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < pointsSize; i += threads * blockDim.x){
        out[i] = (double)d_D[i] + dpxy2[i] <= comp;
    }
}

class CirclesTensorShotgun : public Circles{

private:
    int pointsSize = 0, n = 0, m = 0, k = 0;

    float *px = nullptr, *py = nullptr, *ident = nullptr;
    double *pxy2 = nullptr, *dpxy2 = nullptr;
    bool *doutput;

    hipblasHandle_t handle;
    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;

public:
    __host__ void initialize(const Point *pointsArg, int pointsSizeArg) override {
        pointsSize = pointsSizeArg;

        m = 256;
        n = 256;
        k = 256;
        while(m * k < pointsSize) m *= 2;
        //cout << "Initialized N to: " << n << endl;

        // n -> minimum 64, ideally 256

        px = new float[m * k];
        py = new float[m * n];
        pxy2 = new double[pointsSize];

        for(int i = 0; i < m * k; i++){
            if(i < pointsSize){
                px[i] = (float)pointsArg[i].x;
                py[i] = (float)pointsArg[i].y;
                pxy2[i] = pointsArg[i].x * pointsArg[i].x + pointsArg[i].y * pointsArg[i].y;
            }else{
                px[i] = 0.0f;
                py[i] = 0.0f;
                pxy2[i] = 0.0;
            }
        }

        ident = new float[k * n];
        for(int x = 0; x < n; x++){
            for(int y = 0; y < k; y++){
                ident[y * n + x] = (x == y) ? 1.0f : 0.0f;
            }
        }

        CUBLAS_CHECK(hipblasCreate(&handle));
        CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), m * k * sizeof(d_A[0])));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B), k * n * sizeof(d_B[0])));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C), m * n * sizeof(d_C[0])));

        CUBLAS_CHECK(hipblasSetVector(m * k, sizeof(px[0]), px, 1, d_A, 1));
        CUBLAS_CHECK(hipblasSetVector(k * n, sizeof(ident[0]), ident, 1, d_B, 1));
        CUBLAS_CHECK(hipblasSetVector(n * m, sizeof(py[0]), py, 1, d_C, 1));

        CUDA_CHECK(hipMalloc((void**)&dpxy2, sizeof(double) * pointsSize));
        CUDA_CHECK(hipMemcpy(dpxy2, pxy2, sizeof(double) * pointsSize, hipMemcpyHostToDevice));

        CUDA_CHECK(hipMalloc((void**)&doutput, sizeof(bool) * pointsSize));
    }

    __host__ void run(bool *output, const Point& p1, const Point& p2, const Point& p3) const override {
        const Circumcircle circle(p1, p2, p3);
        const float coefPx = -2.0f * (float)circle.x;
        const float coefPy = -2.0f * (float)circle.y;
        const double comp = circle.r2 - circle.x * circle.x - circle.y * circle.y + 0.0000001;

        //cout << "cpx:  " << coefPx << "  cpy:  " << coefPy << endl;

        CUBLAS_CHECK(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &coefPx,
                                  d_A, HIP_R_32F, k,
                                  d_B, HIP_R_32F, n, &coefPy,
                                  d_C, HIP_R_32F, n,
                                  HIP_R_32F,
                                  CUBLAS_GEMM_DEFAULT_TENSOR_OP));


        //CUBLAS_CHECK(hipblasGetVector(n * n, sizeof(h_D[0]), d_C, 1, h_D, 1));

        /*
        for(int i = 0; i < 10; i++){
            cout << "x:   " << px[i] << "   y:   " << py[i] << endl;
        }

        for(int i = 0; i < 10; i++){
            for(int j = 0; j < 10; j++) cout << h_D[j * n + i] << " ";
            cout << endl;
        }*/

        //for(int i = 0; i < pointsSize; i++){
        //    output[i] = (double)h_D[i] + pxy2[i] <= comp;
        //}

        const int dim = min(max((int)ceil((sqrt((float)pointsSize)) / 32.0f) * 32, 32), 768);

        circlesAssignOutput<<<dim, dim>>>(doutput, d_C, dpxy2, comp, pointsSize, dim);
        CUDA_CHECK(hipMemcpy(output, doutput, sizeof(bool) * pointsSize, hipMemcpyDeviceToHost));
    }

    __host__ void cleanup() override {
        CUBLAS_CHECK(hipblasDestroy(handle));

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        hipFree(dpxy2);
        hipFree(doutput);

        delete[] px;
        delete[] py;
        delete[] pxy2;
        delete[] ident;
    }

    string getFileName() const override {
        return "tensor_shotgun";
    }

};
