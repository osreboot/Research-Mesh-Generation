#include "hip/hip_runtime.h"
#pragma once

#include "math.cuh"

__global__ void circlesDistance(bool *out, const Point* __restrict__ points, const Circumcircle circle, int pointsSize, int threads){
    for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < pointsSize; i += threads){
        out[i] = circle.isInside(points[i]);
    }
}

class CirclesKernelDistance : public Circles{

private:
    const Point *points = nullptr;
    Point *dpoints = nullptr;
    int pointsSize = 0;

public:
    __host__ void initialize(const Point *pointsArg, int pointsSizeArg) override {
        points = pointsArg;
        pointsSize = pointsSizeArg;

        hipMalloc((void**)&dpoints, sizeof(Point) * pointsSize);
        hipMemcpy(dpoints, points, sizeof(Point) * pointsSize, hipMemcpyHostToDevice);
    }

    __host__ void run(bool *output, const Point& p1, const Point& p2, const Point& p3) const override {
        bool *doutput;

        hipMalloc((void**)&doutput, sizeof(bool) * pointsSize);

        const int threads = 128;
        const int blocks = 128 * 128;

        const Circumcircle circle(p1, p2, p3);
        circlesDistance<<<blocks,threads>>>(doutput, dpoints, circle, pointsSize, threads * blocks);
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) cout << "ERROR (CUDA): " << hipGetErrorString(err) << endl;

        hipDeviceSynchronize();

        hipMemcpy(output, doutput, sizeof(bool) * pointsSize, hipMemcpyDeviceToHost);

        hipFree(doutput);
    }

    __host__ void cleanup() override {
        hipFree(dpoints);
    }

    string getFileName() const override {
        return "kernel_distance";
    }

};
