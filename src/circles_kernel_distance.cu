#include "hip/hip_runtime.h"


__global__ void circlesKernelDistance(bool *out, const double* __restrict__ px, const double* __restrict__ py,
                                      const Circumcircle circle, int pointsSize, const int step){
    unsigned int i = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y) * blockDim.x + threadIdx.x;
    for(; i < pointsSize; i += step){
        out[i] = circle.isInside(px[i], py[i]);
    }
}

class CirclesKernelDistance : public Circles{

private:
    const double *px = nullptr, *py = nullptr;
    int pointsSize = 0;
    double *dpx = nullptr, *dpy = nullptr;
    bool *doutput = nullptr;

public:
    __host__ void load(const double *pxArg, const double *pyArg, int pointsSizeArg) override {
        px = pxArg;
        py = pyArg;
        pointsSize = pointsSizeArg;

        CUDA_CHECK(hipMalloc((void**)&dpx, sizeof(double) * pointsSize));
        CUDA_CHECK(hipMalloc((void**)&dpy, sizeof(double) * pointsSize));
        CUDA_CHECK(hipMemcpy(dpx, px, sizeof(double) * pointsSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dpy, py, sizeof(double) * pointsSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc((void**)&doutput, sizeof(bool) * pointsSize));
    }

    __host__ void run(bool *output, const Point& p1, const Point& p2, const Point& p3) override {
        const Circumcircle circle(p1, p2, p3);

        dim3 dimGrid = KERNEL_DIM_GRID;
        dim3 dimBlock = KERNEL_DIM_BLOCK;
        const int step = dimGrid.x * dimGrid.y * dimGrid.z * dimBlock.x;

        circlesKernelDistance<<<dimGrid, dimBlock>>>(doutput, dpx, dpy, circle, pointsSize, step);
        CUDA_CHECK_LAST_ERROR();
    }

    __host__ void save(bool *output) override {
        CUDA_CHECK(hipMemcpy(output, doutput, sizeof(bool) * pointsSize, hipMemcpyDeviceToHost));
    }

    __host__ void cleanup() override {
        hipFree(dpx);
        hipFree(dpy);
        hipFree(doutput);
    }

    string getFileName() const override {
        return "kernel_distance";
    }

};
