#include "hip/hip_runtime.h"
#pragma once

#include "math.cuh"

__global__ void circlesPure(bool *out, const Point* __restrict__ points, const Point p1, const Point p2, const Point p3, int pointsSize, int threads){
    for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < pointsSize; i += threads){
        out[i] = det(p1.x - points[i].x, p1.y - points[i].y, (p1.x * p1.x - points[i].x * points[i].x) + (p1.y * p1.y - points[i].y * points[i].y),
                     p2.x - points[i].x, p2.y - points[i].y, (p2.x * p2.x - points[i].x * points[i].x) + (p2.y * p2.y - points[i].y * points[i].y),
                     p3.x - points[i].x, p3.y - points[i].y, (p3.x * p3.x - points[i].x * points[i].x) + (p3.y * p3.y - points[i].y * points[i].y)) <= 0.00000001;
    }
}

class CirclesKernelPure : public Circles{

private:
    const Point *points = nullptr;
    Point *dpoints = nullptr;
    int pointsSize = 0;

public:
    __host__ void initialize(const Point *pointsArg, int pointsSizeArg) override {
        points = pointsArg;
        pointsSize = pointsSizeArg;

        hipMalloc((void**)&dpoints, sizeof(Point) * pointsSize);
        hipMemcpy(dpoints, points, sizeof(Point) * pointsSize, hipMemcpyHostToDevice);
    }

    __host__ void run(bool *output, const Point& p1, const Point& p2, const Point& p3) const override {
        bool *doutput;

        hipMalloc((void**)&doutput, sizeof(bool) * pointsSize);

        const int threads = 128;
        const int blocks = 128 * 128;

        circlesPure<<<blocks,threads>>>(doutput, dpoints, p1, p2, p3, pointsSize, threads * blocks);
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) cout << "ERROR (CUDA): " << hipGetErrorString(err) << endl;

        hipDeviceSynchronize();

        hipMemcpy(output, doutput, sizeof(bool) * pointsSize, hipMemcpyDeviceToHost);

        hipFree(doutput);
    }

    __host__ void cleanup() override {
        hipFree(dpoints);
    }

    string getFileName() const override {
        return "kernel_pure";
    }

};
