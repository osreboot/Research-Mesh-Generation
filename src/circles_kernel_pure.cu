#include "hip/hip_runtime.h"
#include "math.cuh"

__global__ void circlesKernelPure(bool *out, const double* __restrict__ px, const double* __restrict__ py,
                                  const Point p1, const Point p2, const Point p3, int pointsSize, const int step){
    unsigned int i = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y) * blockDim.x + threadIdx.x;
    for(; i < pointsSize; i += step){
        out[i] = det(p1.x - px[i], p1.y - py[i], (p1.x * p1.x - px[i] * px[i]) + (p1.y * p1.y - py[i] * py[i]),
                     p2.x - px[i], p2.y - py[i], (p2.x * p2.x - px[i] * px[i]) + (p2.y * p2.y - py[i] * py[i]),
                     p3.x - px[i], p3.y - py[i], (p3.x * p3.x - px[i] * px[i]) + (p3.y * p3.y - py[i] * py[i])) <= RADIUS_ERROR;
    }
}

class CirclesKernelPure : public Circles{

private:
    const double *px = nullptr, *py = nullptr;
    int pointsSize = 0;
    double *dpx = nullptr, *dpy = nullptr;
    bool *doutput = nullptr;

public:
    __host__ void load(const double *pxArg, const double *pyArg, int pointsSizeArg) override {
        px = pxArg;
        py = pyArg;
        pointsSize = pointsSizeArg;

        CUDA_CHECK(hipMalloc((void**)&dpx, sizeof(double) * pointsSize));
        CUDA_CHECK(hipMalloc((void**)&dpy, sizeof(double) * pointsSize));
        CUDA_CHECK(hipMemcpy(dpx, px, sizeof(double) * pointsSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dpy, py, sizeof(double) * pointsSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc((void**)&doutput, sizeof(bool) * pointsSize));
    }

    __host__ void run(bool *output, const Point& p1, const Point& p2, const Point& p3) override {
        dim3 dimGrid = KERNEL_DIM_GRID;
        dim3 dimBlock = KERNEL_DIM_BLOCK;
        const int step = dimGrid.x * dimGrid.y * dimGrid.z * dimBlock.x;

        circlesKernelPure<<<dimGrid, dimBlock>>>(doutput, dpx, dpy, p1, p2, p3, pointsSize, step);
        CUDA_CHECK_LAST_ERROR();
    }

    __host__ void save(bool *output) override {
        CUDA_CHECK(hipMemcpy(output, doutput, sizeof(bool) * pointsSize, hipMemcpyDeviceToHost));
    }

    __host__ void cleanup() override {
        hipFree(dpx);
        hipFree(dpy);
        hipFree(doutput);
    }

    string getFileName() const override {
        return "kernel_pure";
    }

};
