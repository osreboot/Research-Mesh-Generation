#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <unordered_set>

#include "primitive.cuh"
#include "math.cuh"
#include "profiler.cuh"

using namespace std;

vector<int> delaunaySearchCircumcircle(const vector<Point>& points, const vector<int>& indices, const Partition& partition, Bounds bounds, Edge edgeActive, int indexP3){
    Triangle triangle = makeClockwise(points,{edgeActive.i1, edgeActive.i2, indexP3});
    const Point p1 = points[triangle.i1];
    const Point p2 = points[triangle.i2];
    const Point p3 = points[triangle.i3];

    // Algorithm source: https://mathworld.wolfram.com/Circumcircle.html
    double a = dets(p1.x, p1.y,
                    p2.x, p2.y,
                    p3.x, p3.y);

    double bx = -dets(p1.x * p1.x + p1.y * p1.y, p1.y,
                      p2.x * p2.x + p2.y * p2.y, p2.y,
                      p3.x * p3.x + p3.y * p3.y, p3.y);
    double by = dets(p1.x * p1.x + p1.y * p1.y, p1.x,
                     p2.x * p2.x + p2.y * p2.y, p2.x,
                     p3.x * p3.x + p3.y * p3.y, p3.x);

    double c = -det(p1.x * p1.x + p1.y * p1.y, p1.x, p1.y,
                    p2.x * p2.x + p2.y * p2.y, p2.x, p2.y,
                    p3.x * p3.x + p3.y * p3.y, p3.x, p3.y);

    double x0 = -bx / (2.0 * a);
    double y0 = -by / (2.0 * a);

    double r = sqrt(bx * bx + by * by - 4.0 * a * c) / (2.0 * abs(a));

    return partition.search((float)x0, (float)y0, (float)r, bounds);
}

int delaunayFindPoint(const vector<Point>& points, const vector<int>& indices, const Partition& partition, int depth, Bounds bounds, Edge edgeActive){
    //float edgeDistance = distance(points[edgeActive.i1], points[edgeActive.i2]);
    float edgeCenterX = (points[edgeActive.i1].x + points[edgeActive.i2].x) / 2.0f;
    float edgeCenterY = (points[edgeActive.i1].y + points[edgeActive.i2].y) / 2.0f;

    profiler::startSection(depth, profiler::L_FIRST);
    int indexP3 = -1;
    for(int offset = 0; offset <= PARTITION_DIMENSION; offset++){
        for(int i : partition.search(edgeCenterX, edgeCenterY, offset, bounds)){
            if(isAboveEdge(points, edgeActive, i)){ // TODO move this test into partition
                if(indexP3 == -1){
                    indexP3 = i;
                }else{
                    Triangle triangle = makeClockwise(points, {edgeActive.i1, edgeActive.i2, indexP3});
                    if(isInCircle(points[triangle.i1], points[triangle.i2], points[triangle.i3], points[i])){
                        indexP3 = i;
                    }
                }
            }
        }
        if(indexP3 != -1) break;
    }
    profiler::stopSection(depth, profiler::L_FIRST);

    if(indexP3 != -1){
        profiler::startSection(depth, profiler::L_CIRCLE);
        bool foundNewPoint;
       // do{
            foundNewPoint = false;
            for(int i : delaunaySearchCircumcircle(points, indices, partition, bounds, edgeActive, indexP3)){
                if(isAboveEdge(points, edgeActive, i)){
                    Triangle triangle = makeClockwise(points, {edgeActive.i1, edgeActive.i2, indexP3});
                    if(isInCircle(points[triangle.i1], points[triangle.i2], points[triangle.i3], points[i])){
                        indexP3 = i;
                        foundNewPoint = true;
                    }
                }
            }
       // }while(foundNewPoint);
        profiler::stopSection(depth, profiler::L_CIRCLE);
    }

    return indexP3;

    /*
    int indexP3 = -1;
    for(int offset = 0; offset <= PARTITION_DIMENSION; offset++){
        for(int i : partition.search(edgeCenterX, edgeCenterY, offset, bounds)){
            if(isAboveEdge(points, edgeActive, i)){
                if(indexP3 == -1){
                    indexP3 = i;
                }else{
                    Triangle triangle = makeClockwise(points, {edgeActive.i1, edgeActive.i2, indexP3});
                    if(isInCircle(points[triangle.i1], points[triangle.i2], points[triangle.i3], points[i])){
                        indexP3 = i;
                    }
                }
            }
        }
    }*/
    return indexP3;
}

// Algorithm source: https://doi.org/10.1016/S0010-4485(97)00082-1
unordered_set<Triangle> delaunay(const vector<Point>& points, const vector<int>& indices, const Partition& partition, Bounds bounds,
                                 unordered_set<Edge> edgesActive, int depth){

    profiler::startBranch(depth);

    // Calculate dividing wall
    profiler::startSection(depth, profiler::WALL);
    Wall wall = Wall::build(bounds, depth);
    profiler::stopSection(depth, profiler::WALL);
    profiler::startSection(depth, profiler::INIT);

    // Divide points by wall side
    vector<int> indicesLeft, indicesRight;
    for(int i : indices){
        if(wall.side(points[i])) indicesRight.push_back(i);
        else indicesLeft.push_back(i);
    }

    unordered_set<Triangle> output;
    unordered_set<Edge> edgesActiveWall, edgesActive1, edgesActive2;

    // Initialize with first edges if we have no active edges
    if(edgesActive.empty() && depth == 0){
        // Find nearest point to dividing line (first edge point)
        int aIndex = indices[0];
        float aDistance = wall.distance(points[indices[0]]);
        for(int i : indices){
            float iDistance = wall.distance(points[i]);
            if(iDistance < aDistance){
                aIndex = i;
                aDistance = iDistance;
            }
        }
        bool aSide = wall.side(points[aIndex]);

        // Find nearest point to A on other side of dividing wall (second edge point)
        int bIndex = aSide ? indicesLeft[0] : indicesRight[0];
        float bDistance = distance(points[aIndex], points[bIndex]);
        for(int i : (aSide ? indicesLeft : indicesRight)){
            float iDistance = distance(points[aIndex], points[i]);
            if(iDistance < bDistance){
                bIndex = i;
                bDistance = iDistance;
            }
        }

        edgesActive.insert({aIndex, bIndex});
        edgesActive.insert({bIndex, aIndex});
    }

    // Divide inherited active edges
    for(Edge edge : edgesActive){
        if(wall.intersects(points[edge.i1], points[edge.i2])){
            edgesActiveWall.insert(edge);
        }else if(wall.side(points[edge.i1])){
            edgesActive2.insert(edge);
        }else edgesActive1.insert(edge);
    }

    profiler::stopSection(depth, profiler::INIT);

    // For all active edges, attempt to complete a triangle and update the active edges list
    while(!edgesActiveWall.empty()){
        Edge edge = *edgesActiveWall.begin();
        edgesActiveWall.erase(edgesActiveWall.begin());

        int p3Index = delaunayFindPoint(points, indices, partition, depth, bounds, edge);

        if(p3Index > -1){ // Check if we've made a new triangle
            profiler::startSection(depth, profiler::SAVE);
            Triangle triangle = makeClockwise(points, {p3Index, edge.i1, edge.i2});
            Triangle triangleOutput = makeSequential(points, triangle);
            if(output.count(triangleOutput)) cerr << "Generated a duplicate triangle!" << endl;
            output.insert(triangleOutput);
            profiler::stopSection(depth, profiler::SAVE);

            // Update active edges based on the new triangle
            profiler::startSection(depth, profiler::CHAIN);
            for(Edge e : vector<Edge>{{triangle.i1, triangle.i2}, {triangle.i3, triangle.i1}}){
                if(wall.intersects(points[e.i1], points[e.i2])){
                    if(edgesActiveWall.count(e) || edgesActiveWall.count(e.reverse())){
                        edgesActiveWall.erase(e);
                        edgesActiveWall.erase(e.reverse());
                    }else edgesActiveWall.insert(e);
                }else if(wall.side(points[e.i1])){
                    if(edgesActive2.count(e) || edgesActive2.count(e.reverse())){
                        edgesActive2.erase(e);
                        edgesActive2.erase(e.reverse());
                    }else edgesActive2.insert(e);
                }else {
                    if (edgesActive1.count(e) || edgesActive1.count(e.reverse())) {
                        edgesActive1.erase(e);
                        edgesActive1.erase(e.reverse());
                    } else edgesActive1.insert(e);
                }
            }
            profiler::stopSection(depth, profiler::CHAIN);
        }
    }

    profiler::stopBranch(depth);

    // Recursively call delaunay triangulation on both sides of the wall
    if(!edgesActive1.empty()){
        Bounds bounds1(bounds.xMin, wall.horizontal ? bounds.xMid : bounds.xMax, bounds.yMin, (!wall.horizontal) ? bounds.yMid : bounds.yMax);
        unordered_set<Triangle> triangles = delaunay(points, indicesLeft, partition, bounds1, edgesActive1, depth + 1);
        profiler::startBranch(depth);
        profiler::startSection(depth, profiler::MERGE);
        output.insert(triangles.begin(), triangles.end());
        profiler::stopSection(depth, profiler::MERGE);
        profiler::stopBranch(depth);
    }
    if(!edgesActive2.empty()){
        Bounds bounds2(wall.horizontal ? bounds.xMid : bounds.xMin, bounds.xMax, (!wall.horizontal) ? bounds.yMid : bounds.yMin, bounds.yMax);
        unordered_set<Triangle> triangles = delaunay(points, indicesRight, partition, bounds2, edgesActive2, depth + 1);
        profiler::startBranch(depth);
        profiler::startSection(depth, profiler::MERGE);
        output.insert(triangles.begin(), triangles.end());
        profiler::stopSection(depth, profiler::MERGE);
        profiler::stopBranch(depth);
    }

    return output;
}

int main(){
    vector<Point> points;

    // Load points from file
    cout << "Loading points from file..." << endl;
    ifstream filePoints("../points.dat");
    if(!filePoints.is_open()) {
        cerr << "Failed to open points file!" << endl;
        return 1;
    }
    string filePointsLine;
    int indexLine = 0;
    while(getline(filePoints, filePointsLine)){
        istringstream iss(filePointsLine);
        Point point = {0.0f, 0.0f};
        iss >> point.x >> point.y;
        points.push_back(point);
        indexLine++;
    }
    filePoints.close();

    // Pre-process input data
    cout << "Partitioning input points..." << endl;
    vector<int> indices;
    Partition partition;
    for(int i = 0; i < points.size(); i++){
        indices.push_back(i);
        partition.insert(points[i], i);
    }

    // Run delaunay triangulation
    cout << "Starting triangulation..." << endl;
    profiler::startProgram();
    unordered_set<Triangle> connections = delaunay(points, indices, partition, Bounds(0.0f, 1.0f, 0.0f, 1.0f), {}, 0);
    profiler::stopProgram();

    // Write connections to file
    cout << "Writing connections to file..." << endl;
    ofstream fileConnections("../connections.dat");
    if(!fileConnections.is_open()){
        cerr << "Failed to open connections file!" << endl;
        return 1;
    }
    for(Triangle triangle : connections){
        fileConnections << triangle.i1 << " " << triangle.i2 << " " << triangle.i3 << " " << endl;
    }
    fileConnections.close();

    return 0;
}
